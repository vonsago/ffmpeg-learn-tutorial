#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define CHECK(call)
{
	const hipError_t error = call;
	if (error != hipSuccess)
	{
		printf("Error: %s:%d, ", __FILE__, __LINE__);
		printf("code:%d, reason: %s\n", error, hipGetErrorString(error));
	}
}


__global__ void sumGPU(float *A, float *B, float *C){
	int i = threadIdx.x;
	printf("calc idx: i: %d\n", i);
	C[i] = A[i]+B[i];
}

int main(int argc, char **argv) {
	float A[3]={1,2,3};
	float B[3]={2,3,4};
	float C[3]={0,0,0};

	float* a_device, *b_device, *c_device;

	hipMalloc((void**)&a_device, 3 * sizeof(float));
    	hipMalloc((void**)&b_device, 3 * sizeof(float));
   	hipMalloc((void**)&c_device, 3 * sizeof(float));

    	hipMemcpy(a_device, A, 3 * sizeof(float), hipMemcpyHostToDevice);
    	hipMemcpy(b_device, B, 3 * sizeof(float), hipMemcpyHostToDevice);

	sumGPU <<<1, 3>>>(a_device, b_device, c_device);

	hipMemcpy(C, c_device, 3 * sizeof(float), hipMemcpyDeviceToHost);

	for(int i=0; i<3; i++)
		printf("%f\n",C[i]);
	hipDeviceReset();
	return (0);
}


#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void doGPU(float * data, float * dst_data){
    int i = threadIdx.x;
    printf("calc idx: i: %d\n", i);
    dst_data[i] = data[i];
}

int doitgpu(float * data, int width, int height, int format, float *ret) {
    float * cuda_data;
    float * dst_data = (float *)malloc(sizeof(data));

    hipMalloc(&cuda_data, sizeof(data));
    hipMemcpy(cuda_data, data, sizeof(data), hipMemcpyHostToDevice);

    doGPU <<<1, sizeof(data)>>>(cuda_data, dst_data);

    hipMemcpy(ret, dst_data, sizeof(data), hipMemcpyDeviceToHost);
    hipDeviceReset();

    return 0;
}
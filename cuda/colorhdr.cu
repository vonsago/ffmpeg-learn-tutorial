#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>


__global__ void doGPU(uint8_t * data, uint8_t * dst_data, int length){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i >= length)
        return;
    dst_data[i] = data[i];
}

int doitgpu(uint8_t * data, int width, int height, int format, uint8_t *ret, int length) {
    uint8_t * cuda_data;
    uint8_t * dst_data;

    hipMalloc(&cuda_data, length);
    hipMalloc(&dst_data, length);
    hipMemcpy(cuda_data, data, length, hipMemcpyHostToDevice);

    int blocks = (length + 128-1)/128;
    int threads = 128;
    doGPU <<<blocks, threads>>>(cuda_data, dst_data, length);

    hipMemcpy(ret, dst_data, length, hipMemcpyDeviceToHost);
    hipDeviceReset();

    return 0;
}
